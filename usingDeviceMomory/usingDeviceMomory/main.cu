#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void)
{
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void)
{
	int* dDataPtr;
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
	printf("hipMalloc = %s\n", hipGetErrorName(errorCode));

	checkDeviceMemory();

	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();
}