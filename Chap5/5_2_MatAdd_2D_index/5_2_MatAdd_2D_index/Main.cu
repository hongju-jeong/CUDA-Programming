#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ROW_SIZE 32
#define COL_SIZE 32

__global__ void matAdd_2D_index(float* _dA, float* _dB, float* _dC)
{
	unsigned int col = threadIdx.x;
	unsigned int row = threadIdx.y;
	unsigned int index = row * blockDim.x + col;

	_dC[index] = _dA[index] + _dB[index];
}

int main(void) {

	// set the matrix on host
	float A[ROW_SIZE][COL_SIZE] = { 0 };
	float B[ROW_SIZE][COL_SIZE] = { 0 };
	float C[ROW_SIZE][COL_SIZE] = { 0 };
	float hC[ROW_SIZE][COL_SIZE] = { 0 };

	// set memory space on the device memory
	int matSize = ROW_SIZE * COL_SIZE;
	float* dA = NULL;
	float* dB = NULL;
	float* dC = NULL;
	hipMalloc(&dA, sizeof(float) * matSize); hipMemset(dA, 0, sizeof(float) * matSize);
	hipMalloc(&dB, sizeof(float) * matSize); hipMemset(dB, 0, sizeof(float) * matSize);
	hipMalloc(&dC, sizeof(float) * matSize); hipMemset(dC, 0, sizeof(float) * matSize);

	for (int iRow = 0; iRow < ROW_SIZE; iRow++)
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			A[iRow][iCol] = rand() % 100;
			B[iRow][iCol] = rand() % 100;
			C[iRow][iCol] = A[iRow][iCol] + B[iRow][iCol];
		}

	hipMemcpy(dA, A, sizeof(float) * matSize, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * matSize, hipMemcpyHostToDevice);

	dim3 blockDim(COL_SIZE, ROW_SIZE);
	matAdd_2D_index << <1, blockDim >> > (dA, dB, dC);

	hipMemcpy(hC, dC, sizeof(float) * matSize, hipMemcpyDeviceToHost);

	bool isCorrect = true;
	for (int iRow= 0 ; iRow < ROW_SIZE; iRow++)
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			if (hC[iRow][iCol] != C[iRow][iCol]) {
				isCorrect = false;
				break;
			}
		}

	if (isCorrect) printf("GPU works well!\n");
	else printf("GPU fail to make correct resutl(s)..\n");


	return 0;
}